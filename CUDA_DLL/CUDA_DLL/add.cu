#include "hip/hip_runtime.h"
#include "add.cuh"


/**
* CUDA kernel
*/
__global__ void kernel(int* a, int* b, int* c) {

	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

/**
* Function adds two numbers and stores the result in c
*/
void addTwoNum(int* a, int* b, int* c) {

	int* d_a, * d_b, * d_c;

	hipMalloc((void**)&d_a, sizeof(int));
	hipMalloc((void**)&d_b, sizeof(int));
	hipMalloc((void**)&d_c, sizeof(int));

	hipMemcpy(d_a, a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, sizeof(int), hipMemcpyHostToDevice);

	kernel << <1, 1 >> > (d_a, d_b, d_c);

	hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}